#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>

// the block size is 1<<LOG_BK_SIZE in x and y direction
#define LOG_BK_SIZE (5)



//do the cphase gate. set a bit in mask to 1 if you want the corresponding qbit to 
//partake in the cphase.
//no shared memory needed
//run in a 2d grid that stretches over dm10
__global__ void cphase(float *dm10, unsigned int mask) {
    const int x = (blockIdx.x << LOG_BK_SIZE) + threadIdx.x;
    const int y = (blockIdx.y << LOG_BK_SIZE) + threadIdx.y;

    //if exactly one of x,y has all bits in mask set
    if ((x & mask == mask) != (y & mask == mask)) {
        dm10[(x << 10) + y] = -dm10[(x << 10) + y];
    }
}


//do the hadamard on a qbit
//this needs to be run on a grid over d10
//mask must have exactly one bit flipped, denoting which byte is involved
//the results is multiplied with mul, to obtain unitary, set mul = 1/sqrt(2)

__global__ void hadamard(float *dm10, unsigned int mask, float mul) { 

    const int x = (blockIdx.x << LOG_BK_SIZE) + threadIdx.x;
    const int y = (blockIdx.y << LOG_BK_SIZE) + threadIdx.y;

    if (x>y) return;
    if ((x & mask) || (y & mask)) return;


    //TODO: optimize to only read once and write once...

    float *a_re = &dm10[ (x << 10) + y ];
    float *a_im = &dm10[ x + (y << 10) ];

    //note that y|mask >= y, so we know which one is the real part
    float *b_re = &dm10[ (x << 10) + (y | mask) ];
    float *b_im = &dm10[ x + ((y | mask) << 10) ];

    float *c_re = &dm10[ ((x | mask) << 10) + (y | mask) ];
    float *c_im = &dm10[ (x | mask) + ((y | mask) << 10) ];


    float new_a_re = *a_re + *c_re + 2*(*b_re);
    float new_a_im = *a_im + *c_im;
    *a_re = mul*new_a_re;
    *a_im = mul*new_a_im;
    float new_b_re = *a_re - *c_re;
    float new_b_im = *a_im - *c_im - 2*(*b_im);
    *b_re = mul*new_b_re;
    *b_im = mul*new_b_im;
    float new_c_re = *a_re + *c_re - 2*(*b_re);
    float new_c_im = *a_im + *c_im;
    *c_re = mul*new_c_re;
    *c_im = mul*new_c_im;
}



