#include <hip/hip_runtime.h> 
#include <hip/hip_complex.h>

// the block size is 1<<LOG_BK_SIZE in x and y direction
#define LOG_BK_SIZE (5)

//this defines the scheme by which the density matrix is stored in a dm10
//we always require x <= y for the real part and
#define ADDR_BARE(x,y,ri_flag,n) (((((x)<<(n)) | (y)) << 1) | (ri_flag))
#define ADDR_TRIU(x,y,ri_flag,n) (((x) <= (y)) ? ADDR_BARE(x,y,ri_flag,n) : ADDR_BARE(y,x,ri_flag,n))
#define ADDR_REAL(x,y,n) ADDR_TRIU(x,y,0,n)
#define ADDR_IMAG(x,y,n) ADDR_TRIU(x,y,1,n)



//do the cphase gate. set a bit in mask to 1 if you want the corresponding qubit to 
//partake in the cphase.
//run in a 2d grid that stretches over dm10
__global__ void cphase(double *dm10, unsigned int mask, unsigned int no_qubits) {
    const int x = (blockIdx.x << LOG_BK_SIZE) + threadIdx.x;
    const int y = (blockIdx.y << LOG_BK_SIZE) + threadIdx.y;

    //if exactly one of x,y has all bits in mask set
    if (((x & mask) == mask) != ((y & mask) == mask)) {

        if (x <= y) { //real part
            dm10[ADDR_REAL(x,y,no_qubits)] = -dm10[ADDR_REAL(x,y,no_qubits)];
        }
        else { //imaginary part
            dm10[ADDR_IMAG(y,x,no_qubits)] = -dm10[ADDR_IMAG(y,x,no_qubits)];
        }
    }
}


//do the hadamard on a qubit
//mask must have exactly one bit flipped, denoting which byte is involved
//the results is multiplied with mul, to obtain trace preserving map, set mul = 0.5
__global__ void hadamard(double *dm10, unsigned int mask, double mul, unsigned int no_qubits) { 

    int x = (blockIdx.x << LOG_BK_SIZE) + threadIdx.x;
    int y = (blockIdx.y << LOG_BK_SIZE) + threadIdx.y;

    if((x&mask) && (~y&mask)) { //real part
        x = x & ~mask;
        if (x <= y) {
            double a = dm10[ADDR_REAL(x, y, no_qubits)];
            double b = dm10[ADDR_REAL(x|mask, y, no_qubits)];
            double c = dm10[ADDR_REAL(x, y|mask, no_qubits)];
            double d = dm10[ADDR_REAL(x|mask, y|mask, no_qubits)];

            double new_a = a+b+c+d;
            double new_b = a-b+c-d;
            double new_c = a+b-c-d;
            double new_d = a-b-c+d;

            dm10[ADDR_REAL(x, y, no_qubits)] = mul*new_a;
            dm10[ADDR_REAL(x|mask, y, no_qubits)] = mul*new_b;
            dm10[ADDR_REAL(x, y|mask, no_qubits)] = mul*new_c;
            dm10[ADDR_REAL(x|mask, y|mask, no_qubits)] = mul*new_d;
        }
    }
    if ((~x&mask) && (y&mask)) { //do the imaginary part
        y = y & ~mask;
        if (y <= x){
            double a = dm10[ADDR_IMAG(y, x, no_qubits)];
            double b = dm10[ADDR_IMAG(y|mask, x, no_qubits)];
            double c = dm10[ADDR_IMAG(y, x|mask, no_qubits)];
            double d = dm10[ADDR_IMAG(y|mask, x|mask, no_qubits)];

            double new_a = a+b+c+d;
            double new_b = a-b+c-d;
            double new_c = a+b-c-d;
            double new_d = a-b-c+d;

            dm10[ADDR_IMAG(y, x, no_qubits)] = mul*new_a;
            dm10[ADDR_IMAG(y|mask, x, no_qubits)] = mul*new_b;
            dm10[ADDR_IMAG(y, x|mask, no_qubits)] = mul*new_c;
            dm10[ADDR_IMAG(y|mask, x|mask, no_qubits)] = mul*new_d;
        }
    }
}


// amplitude damping and phase damping on one qubit
// mask: bit mask selecting the qbit
// gamma = probability for amplitude decay, i.e gamma = 1-exp(-t/T1)
// s1mgamma = sqrt(1-gamma)
// s1mlambda = sqrt(1-lambda), where lambda is the probability for a phase flip, i.e. lambda = 1- exp(-t/T2)
__global__ void amp_ph_damping(double *dm10, unsigned int mask, double gamma, double s1mgamma, double s1mlambda, unsigned int no_qubits) {

    int x = (blockIdx.x << LOG_BK_SIZE) + threadIdx.x;
    int y = (blockIdx.y << LOG_BK_SIZE) + threadIdx.y;

    int ri_flag = 1;

    if (x >= y) ri_flag = 0;


    double f = dm10[ADDR_TRIU(x, y, ri_flag, no_qubits)];

    if (x&y&mask) { //c block
        dm10[ADDR_TRIU(x^mask,y^mask,ri_flag, no_qubits)]  += gamma * f;
        f = f - gamma*f;
    } 
    else if ((~x)&(~y)&mask) {
        return;
    }
    else { // b block
        f *= s1mgamma * s1mlambda;
    }

    dm10[ADDR_TRIU(x, y, ri_flag, no_qubits)] = f;
}

//copy the two diagonal blocks of one ancilla into reduced density matrices
//multiply the two with two numbers (inverse of the traces for instance, to implement measurement)
//note that because major bit banging is required to figure out the new adresses,
//the qubit index is passed as an integer, not as a bitmask!

__global__ void dm_reduce(double *dm10, unsigned int bit_idx, double *dm9_0, double *dm9_1, double mul0, double mul1, unsigned int no_qubits) {
    int x = (blockIdx.x << LOG_BK_SIZE) + threadIdx.x;
    int y = (blockIdx.y << LOG_BK_SIZE) + threadIdx.y;

    int ri_flag = 1;
    if (x >= y) ri_flag = 0;

    unsigned int mask = (1 << bit_idx);                  // e.g. 00010000 if bit_idx == 4

    int block; //0 or 1?
    
    if(x&y&mask) block = 1;
    else if((~x)&(~y)&mask) block = 0;
    else return;

    unsigned int lower_mask =  (1 << bit_idx) - 1;       // e.g. 00001111
    unsigned int upper_mask = ~((1 << (bit_idx+1)) - 1); // e.g. 11100000

    unsigned int x9, y9; //new adresses

    x9 =  ((x&upper_mask) >> 1) | (x & lower_mask);
    y9 =  ((y&upper_mask) >> 1) | (y & lower_mask);

    
    if(block == 0) {
        dm9_0[ADDR_TRIU(x9, y9, ri_flag, no_qubits-1)] = mul0*dm10[ADDR_TRIU(x,y,ri_flag, no_qubits)];
    }
    if(block == 1) {
        dm9_1[ADDR_TRIU(x9, y9, ri_flag, no_qubits-1)] = mul1*dm10[ADDR_TRIU(x,y,ri_flag, no_qubits)];
    }
}



//trace kernel
//copy the diagonal elements to out, in order to do effective 
//calculation of subtraces.
//run over a 1x9 grid!
__global__ void get_diag(double *dm9, double *out, unsigned int no_qubits) {
    int x = (blockIdx.x <<  LOG_BK_SIZE) + threadIdx.x;
    out[x] = dm9[ADDR_BARE(x,x,0,no_qubits)];
}

//inverse of dm_reduce
//run over 9x9 grid!
__global__ void dm_inflate(double *dm10, unsigned int bit_idx, double *dm9_0, double *dm9_1, unsigned int no_qubits) {
    int x9 = (blockIdx.x << LOG_BK_SIZE) + threadIdx.x;
    int y9 = (blockIdx.y << LOG_BK_SIZE) + threadIdx.y;

    int ri_flag = 1;
    if (x9 >= y9) ri_flag = 0;

    unsigned int mask = (1 << bit_idx);                  // e.g. 00010000 if bit_idx == 4

    unsigned int lower_mask =  (1 << bit_idx) - 1;       // e.g. 00001111
    unsigned int upper_mask = ~((1 << bit_idx) - 1);     // e.g. 11110000

    //calculate new adresses
    unsigned int x, y;

    x =  ((x9&upper_mask) << 1) | (x9 & lower_mask);
    y =  ((y9&upper_mask) << 1) | (y9 & lower_mask);

    dm10[ADDR_TRIU(x, y, ri_flag, no_qubits)] = dm9_0[ADDR_TRIU(x9, y9, ri_flag, no_qubits-1)];
    dm10[ADDR_TRIU(x|mask, y|mask, ri_flag, no_qubits)] = dm9_1[ADDR_TRIU(x9, y9, ri_flag, no_qubits-1)];
}
